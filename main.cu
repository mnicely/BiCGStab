#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <cctype>
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <stdexcept>
#include <string>
#include <typeinfo>  // for usage of C++ typeid

#include "hipblas.h"
#include "hipsparse.h"
#include <hip/hip_runtime.h>
//  #include "helper_cusolver.h"
#include "mmio.h"

#include "mmio_wrapper.h"

//  #include "hip/hip_runtime_api.h"

// profiling the code
#define TIME_INDIVIDUAL_LIBRARY_CALLS

#define DBICGSTAB_MAX_ULP_ERR 100
#define DBICGSTAB_EPS 1.E-14f  // 9e-2

#ifndef CUDA_RT_CALL
#define CUDA_RT_CALL( call )                                                                                           \
    {                                                                                                                  \
        auto        status = static_cast<hipError_t>( call );                                                         \
        auto        format = "ERROR: CUDA_RT call \"%s\" in line %d of file %s failed with code (%d).\n";              \
        auto        size   = std::snprintf( nullptr, 0, format, #call, __LINE__, __FILE__, status );                   \
        std::string output( size + 1, '\0' );                                                                          \
        std::sprintf( &output[0], format, #call, __LINE__, __FILE__, status );                                         \
        if ( status != hipSuccess )                                                                                   \
            throw std::runtime_error( output );                                                                        \
    }
#endif  // CUDA_RT_CALL

#ifndef CUBLAS_CALL
#define CUBLAS_CALL( call )                                                                                            \
    {                                                                                                                  \
        auto        status = static_cast<hipblasStatus_t>( call );                                                      \
        auto        format = "ERROR: CULBAS call \"%s\" in line %d of file %s failed with code (%d).\n";               \
        auto        size   = std::snprintf( nullptr, 0, format, #call, __LINE__, __FILE__, status );                   \
        std::string output( size + 1, '\0' );                                                                          \
        std::sprintf( &output[0], format, #call, __LINE__, __FILE__, status );                                         \
        if ( status != HIPBLAS_STATUS_SUCCESS )                                                                         \
            throw std::runtime_error( output );                                                                        \
    }
#endif  // CUBLAS_CALL

#ifndef CUSPARSE_CALL
#define CUSPARSE_CALL( call )                                                                                          \
    {                                                                                                                  \
        auto        status = static_cast<hipsparseStatus_t>( call );                                                    \
        auto        format = "ERROR: CUSPARSE call \"%s\" in line %d of file %s failed with code (%d).\n";             \
        auto        size   = std::snprintf( nullptr, 0, format, #call, __LINE__, __FILE__, status );                   \
        std::string output( size + 1, '\0' );                                                                          \
        std::sprintf( &output[0], format, #call, __LINE__, __FILE__, status );                                         \
        if ( status != HIPSPARSE_STATUS_SUCCESS )                                                                       \
            throw std::runtime_error( output );                                                                        \
    }
#endif  // CUSPARSE_CALL

#define CLEANUP( )                                                                                                     \
    do {                                                                                                               \
        if ( x )                                                                                                       \
            free( x );                                                                                                 \
        if ( f )                                                                                                       \
            free( f );                                                                                                 \
        if ( r )                                                                                                       \
            free( r );                                                                                                 \
        if ( rw )                                                                                                      \
            free( rw );                                                                                                \
        if ( p )                                                                                                       \
            free( p );                                                                                                 \
        if ( pw )                                                                                                      \
            free( pw );                                                                                                \
        if ( s )                                                                                                       \
            free( s );                                                                                                 \
        if ( t )                                                                                                       \
            free( t );                                                                                                 \
        if ( v )                                                                                                       \
            free( v );                                                                                                 \
        if ( tx )                                                                                                      \
            free( tx );                                                                                                \
        if ( Aval )                                                                                                    \
            free( Aval );                                                                                              \
        if ( AcolsIndex )                                                                                              \
            free( AcolsIndex );                                                                                        \
        if ( ArowsIndex )                                                                                              \
            free( ArowsIndex );                                                                                        \
        if ( Mval )                                                                                                    \
            free( Mval );                                                                                              \
        if ( devPtrX )                                                                                                 \
            CUDA_RT_CALL( hipFree( devPtrX ) );                                                                       \
        if ( devPtrF )                                                                                                 \
            CUDA_RT_CALL( hipFree( devPtrF ) );                                                                       \
        if ( devPtrR )                                                                                                 \
            CUDA_RT_CALL( hipFree( devPtrR ) );                                                                       \
        if ( devPtrRW )                                                                                                \
            CUDA_RT_CALL( hipFree( devPtrRW ) );                                                                      \
        if ( devPtrP )                                                                                                 \
            CUDA_RT_CALL( hipFree( devPtrP ) );                                                                       \
        if ( devPtrS )                                                                                                 \
            CUDA_RT_CALL( hipFree( devPtrS ) );                                                                       \
        if ( devPtrT )                                                                                                 \
            CUDA_RT_CALL( hipFree( devPtrT ) );                                                                       \
        if ( devPtrV )                                                                                                 \
            CUDA_RT_CALL( hipFree( devPtrV ) );                                                                       \
        if ( devPtrAval )                                                                                              \
            CUDA_RT_CALL( hipFree( devPtrAval ) );                                                                    \
        if ( devPtrAcolsIndex )                                                                                        \
            CUDA_RT_CALL( hipFree( devPtrAcolsIndex ) );                                                              \
        if ( devPtrArowsIndex )                                                                                        \
            CUDA_RT_CALL( hipFree( devPtrArowsIndex ) );                                                              \
        if ( devPtrMval )                                                                                              \
            CUDA_RT_CALL( hipFree( devPtrMval ) );                                                                    \
        if ( stream )                                                                                                  \
            CUDA_RT_CALL( hipStreamDestroy( stream ) );                                                               \
        if ( cublasHandle )                                                                                            \
            CUBLAS_CALL( hipblasDestroy( cublasHandle ) );                                                              \
        if ( cusparseHandle )                                                                                          \
            CUSPARSE_CALL( hipsparseDestroy( cusparseHandle ) );                                                        \
        fflush( stdout );                                                                                              \
    } while ( 0 )

#ifndef STRCPY
#define STRCPY( sFilePath, nLength, sPath ) strcpy( sFilePath, sPath )
#endif

#ifndef FOPEN
#define FOPEN( fHandle, filename, mode ) ( fHandle = fopen( filename, mode ) )
#endif

// PULLED from hip/hip_runtime_api.h
inline char *sdkFindFilePath( const char *filename, const char *executable_path ) {
    // <executable_name> defines a variable that is replaced with the name of the
    // executable

    const char *searchPath[] = {
        "./",   // same dir
        "../",  // upper dir
    };

    // Extract the executable name
    std::string executable_name;

    if ( executable_path != 0 ) {
        executable_name = std::string( executable_path );

        // Linux & OSX path delimiter
        size_t delimiter_pos = executable_name.find_last_of( '/' );
        executable_name.erase( 0, delimiter_pos + 1 );
    }

    // Loop over all search paths and return the first hit
    for ( unsigned int i = 0; i < sizeof( searchPath ) / sizeof( char * ); ++i ) {
        std::string path( searchPath[i] );
        size_t      executable_name_pos = path.find( "<executable_name>" );

        // If there is executable_name variable in the searchPath
        // replace it with the value
        if ( executable_name_pos != std::string::npos ) {
            if ( executable_path != 0 ) {
                path.replace( executable_name_pos, strlen( "<executable_name>" ), executable_name );
            } else {
                // Skip this path entry if no executable argument is given
                continue;
            }
        }

        // Test if the file exists
        path.append( filename );
        FILE *fp;
        FOPEN( fp, path.c_str( ), "rb" );

        if ( fp != NULL ) {
            fclose( fp );
            // File found
            // returning an allocated array here for backwards compatibility reasons
            char *file_path = reinterpret_cast<char *>( malloc( path.length( ) + 1 ) );
            STRCPY( file_path, path.length( ) + 1, path.c_str( ) );
            return file_path;
        }

        if ( fp ) {
            fclose( fp );
        }
    }

    // File not found
    return 0;
}

// #include <stddef.h>
// #include <sys/resource.h>
// #include <sys/sysctl.h>
// #include <sys/time.h>
// #include <sys/types.h>
// double second( void ) {
//     struct timeval tv;
//     gettimeofday( &tv, NULL );
//     return ( double )tv.tv_sec + ( double )tv.tv_usec / 1000000.0;
// }

static void gpu_pbicgstab(
    hipblasHandle_t             cublasHandle,
    hipsparseHandle_t           cusparseHandle,
    int                        m,
    int                        n,
    int                        nnz,
    const hipsparseSpMatDescr_t descra, /* the coefficient matrix in CSR format */
    double *                   a,
    const int *                ia,
    const int *                ja,
    const hipsparseSpMatDescr_t descrm, /* the preconditioner in CSR format, lower & upper triangular factor */
    double *                   vm,
    const int *                im,
    const int *                jm,
    const hipsparseMatDescr_t   descra_uli,
    hipsparseDnVecDescr_t       vecX,
    hipsparseDnVecDescr_t       vecY,
    hipsparseSpSVDescr_t        spsvDescr_l,
    hipsparseSpSVDescr_t        spsvDescr_u,
    hipsparseFillMode_t         fillmode,
    hipsparseDiagType_t         diagtype,
    csrilu02Info_t             info_M,
    //    csrsv2Info_t             info_l,
    //    csrsv2Info_t             info_u, /* the analysis of the lower and upper triangular parts */
    void *      dBuffer,
    void *      dBuffer_ilu,
    double *    f,
    double *    r,
    double *    rw,
    double *    p,
    double *    pw,
    double *    s,
    double *    t,
    double *    v,
    double *    x,
    int         maxit,
    double      tol,
    double      ttt_sv,
    hipEvent_t ttm,
    hipEvent_t ttm2,
    float       ttm_ttm2,
    hipEvent_t ttl,
    hipEvent_t ttl2,
    float       ttl_ttl2,
    hipEvent_t ttu,
    hipEvent_t ttu2,
    float       ttu_ttu2 ) {

    double rho        = 0.0;
    double rhop       = 0.0;
    double beta       = 0.0;
    double alpha      = 0.0;
    double negalpha   = 0.0;
    double omega      = 0.0;
    double negomega   = 0.0;
    double temp       = 0.0;
    double temp2      = 0.0;
    double nrmr       = 0.0;
    double nrmr0      = 0.0;
    rho               = 0.0;
    const double zero = 0.0;
    const double one  = 1.0;
    double       mone = -1.0;
    int          i    = 0;
    int          j    = 0;
    // double       ttl, ttl2, ttu, ttu2, ttm, ttm2;
    // double ttt_mv = 0.0;

// WARNING: Analysis is done outside of the function (and the time taken by it is passed to the function in variable
// ttt_sv)

// compute initial residual r0=b-Ax0 (using initial guess in x)
#ifdef TIME_INDIVIDUAL_LIBRARY_CALLS
    // CUDA_RT_CALL( hipDeviceSynchronize( ) );
    // ttm = second( );
    double ttt_mv = 0.0;
    CUDA_RT_CALL( hipEventRecord( ttm ) );
#endif

    void * dBuffer_mv    = NULL;
    size_t bufferSize_mv = 0;

    // CUSPARSE_CALL( hipsparseDcsrmv(
    //     cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &one, descra, a, ia, ja, x, &zero, r ) );

    CUSPARSE_CALL( hipsparseCreateDnVec( &vecX, n, x, HIP_R_64F ) )
    CUSPARSE_CALL( hipsparseCreateDnVec( &vecY, n, r, HIP_R_64F ) )

    // allocate an external buffer if needed
    CUSPARSE_CALL( hipsparseSpMV_bufferSize( cusparseHandle,
                                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                            &one,
                                            descra,
                                            vecX,
                                            &zero,
                                            vecY,
                                            HIP_R_64F,
                                            HIPSPARSE_MV_ALG_DEFAULT,
                                            &bufferSize_mv ) )

    CUDA_RT_CALL( hipMalloc( &dBuffer_mv, bufferSize_mv ) )

    // execute SpMV
    CUSPARSE_CALL( hipsparseSpMV( cusparseHandle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &one,
                                 descra,
                                 vecX,
                                 &zero,
                                 vecY,
                                 HIP_R_64F,
                                 HIPSPARSE_MV_ALG_DEFAULT,
                                 dBuffer_mv ) )

#ifdef TIME_INDIVIDUAL_LIBRARY_CALLS
    // CUDA_RT_CALL( hipDeviceSynchronize( ) );
    CUDA_RT_CALL( hipEventRecord( ttm2 ) );
    CUDA_RT_CALL( hipEventSynchronize( ttm2 ) );
    CUDA_RT_CALL( hipEventElapsedTime( &ttm_ttm2, ttm, ttm2 ) );
    // ttm2 = second( );
    ttt_mv += ( ttm_ttm2 * 1e-3 );
    printf( "matvec %f (s)\n", ttm_ttm2 * 1e-3 );
#endif
    // 	 CUBLAS_CALL(hipblasDscal(cublasHandle, n, &mone, r, 1));
    // 	 CUBLAS_CALL(hipblasDaxpy(cublasHandle, n, &one, f, 1, r, 1));
    // copy residual r into r^{\hat} and p
    // 	 CUBLAS_CALL(hipblasDcopy(cublasHandle, n, r, 1, rw, 1));
    // 	 CUBLAS_CALL(hipblasDcopy(cublasHandle, n, r, 1, p, 1));
    // 	 CUBLAS_CALL(hipblasDnrm2(cublasHandle, n, r, 1, &nrmr0));

    CUBLAS_CALL( hipblasScalEx( cublasHandle, n, &mone, HIP_R_64F, r, HIP_R_64F, 1, HIP_R_64F ) );
    CUBLAS_CALL( hipblasAxpyEx( cublasHandle, n, &one, HIP_R_64F, f, HIP_R_64F, 1, r, HIP_R_64F, 1, HIP_R_64F ) );
    // copy residual r into r^{\hat} and p
    CUBLAS_CALL( hipblasDcopy( cublasHandle, n, r, 1, rw, 1 ) );
    CUBLAS_CALL( hipblasDcopy( cublasHandle, n, r, 1, p, 1 ) );
    CUBLAS_CALL( hipblasDnrm2( cublasHandle, n, r, 1, &nrmr0 ) );
    printf( "gpu, init residual:norm %20.16f\n", nrmr0 );

    for ( i = 0; i < maxit; ) {
        rhop = rho;
        // 		 CUBLAS_CALL(hipblasDdot(cublasHandle, n, rw, 1, r, 1, &rho));
        CUBLAS_CALL(
            hipblasDotEx( cublasHandle, n, rw, HIP_R_64F, 1, r, HIP_R_64F, 1, &rho, HIP_R_64F, HIP_R_64F ) );

        if ( i > 0 ) {
            beta     = ( rho / rhop ) * ( alpha / omega );
            negomega = -omega;
            // 			 CUBLAS_CALL(hipblasDaxpy(cublasHandle,n, &negomega, v, 1, p, 1));
            // 			 CUBLAS_CALL(hipblasDscal(cublasHandle,n, &beta, p, 1));
            // 			 CUBLAS_CALL(hipblasDaxpy(cublasHandle,n, &one, r, 1, p, 1));
            CUBLAS_CALL( hipblasAxpyEx(
                cublasHandle, n, &negomega, HIP_R_64F, v, HIP_R_64F, 1, p, HIP_R_64F, 1, HIP_R_64F ) );
            CUBLAS_CALL( hipblasScalEx( cublasHandle, n, &beta, HIP_R_64F, p, HIP_R_64F, 1, HIP_R_64F ) );
            CUBLAS_CALL(
                hipblasAxpyEx( cublasHandle, n, &one, HIP_R_64F, r, HIP_R_64F, 1, p, HIP_R_64F, 1, HIP_R_64F ) );
        }
        // preconditioning step (lower and upper triangular solve)
#ifdef TIME_INDIVIDUAL_LIBRARY_CALLS
        // CUDA_RT_CALL( hipDeviceSynchronize( ) );
        // ttl = second( );
        CUDA_RT_CALL( hipEventRecord( ttl ) );
#endif
        // CUSPARSE_CALL( hipsparseSetMatFillMode( descrm, HIPSPARSE_FILL_MODE_LOWER ) );
        // CUSPARSE_CALL( hipsparseSetMatDiagType( descrm, HIPSPARSE_DIAG_TYPE_UNIT ) );
        // // 		 CUSPARSE_CALL(cusparseDcsrsv_solve(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE,n,&one,
        // // descrm,vm,im,jm,info_l,p,t));
        // CUSPARSE_CALL( hipsparseDcsrsv2_solve( cusparseHandle,
        //                                       HIPSPARSE_OPERATION_NON_TRANSPOSE,
        //                                       n,
        //                                       nnz,
        //                                       &one,
        //                                       descrm,
        //                                       vm,
        //                                       im,
        //                                       jm,
        //                                       info_l,
        //                                       p,
        //                                       t,
        //                                       HIPSPARSE_SOLVE_POLICY_NO_LEVEL,
        //                                       dBuffer_l ) );

        CUSPARSE_CALL( hipsparseCreateDnVec( &vecX, n, p, HIP_R_64F ) )
        CUSPARSE_CALL( hipsparseCreateDnVec( &vecY, n, t, HIP_R_64F ) )

        // execute SpSV Lower
        CUSPARSE_CALL( hipsparseSpSV_solve( cusparseHandle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &one,
                                           descrm,
                                           vecX,
                                           vecY,
                                           HIP_R_64F,
                                           HIPSPARSE_SPSV_ALG_DEFAULT,
                                           spsvDescr_l ) )

#ifdef TIME_INDIVIDUAL_LIBRARY_CALLS
        // CUDA_RT_CALL( hipDeviceSynchronize( ) );
        CUDA_RT_CALL( hipEventRecord( ttl2 ) );
        CUDA_RT_CALL( hipEventSynchronize( ttl2 ) );
        CUDA_RT_CALL( hipEventElapsedTime( &ttl_ttl2, ttl, ttl2 ) );

        // ttl2 = second( );
        // ttu  = second( );
        CUDA_RT_CALL( hipEventRecord( ttu ) );

#endif
        // CUSPARSE_CALL( hipsparseSetMatFillMode( descrm, HIPSPARSE_FILL_MODE_UPPER ) );
        // CUSPARSE_CALL( hipsparseSetMatDiagType( descrm, HIPSPARSE_DIAG_TYPE_NON_UNIT ) );
        // // 		 CUSPARSE_CALL(cusparseDcsrsv_solve(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE,n,&one,
        // // descrm,vm,im,jm,info_u,t,pw));
        // CUSPARSE_CALL( hipsparseDcsrsv2_solve( cusparseHandle,
        //                                       HIPSPARSE_OPERATION_NON_TRANSPOSE,
        //                                       n,
        //                                       nnz,
        //                                       &one,
        //                                       descrm,
        //                                       vm,
        //                                       im,
        //                                       jm,
        //                                       info_u,
        //                                       t,
        //                                       pw,
        //                                       HIPSPARSE_SOLVE_POLICY_NO_LEVEL,
        //                                       dBuffer_u ) );

        CUSPARSE_CALL( hipsparseCreateDnVec( &vecX, n, t, HIP_R_64F ) )
        CUSPARSE_CALL( hipsparseCreateDnVec( &vecY, n, pw, HIP_R_64F ) )

        // execute SpSV Lower
        CUSPARSE_CALL( hipsparseSpSV_solve( cusparseHandle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &one,
                                           descrm,
                                           vecX,
                                           vecY,
                                           HIP_R_64F,
                                           HIPSPARSE_SPSV_ALG_DEFAULT,
                                           spsvDescr_u ) )

#ifdef TIME_INDIVIDUAL_LIBRARY_CALLS
        // CUDA_RT_CALL( hipDeviceSynchronize( ) );
        // ttu2 = second( );
        CUDA_RT_CALL( hipEventRecord( ttu2 ) );
        CUDA_RT_CALL( hipEventSynchronize( ttu2 ) );
        CUDA_RT_CALL( hipEventElapsedTime( &ttu_ttu2, ttu, ttu2 ) );
        ttt_sv += ( ttl_ttl2 * 1e-3 ) + ( ttu_ttu2 * 1e-3 );
        printf( "solve lower %f (s), upper %f (s) \n", ttl_ttl2 * 1e-3, ttu_ttu2 * 1e-3 );
#endif

        // matrix-vector multiplication
#ifdef TIME_INDIVIDUAL_LIBRARY_CALLS
        // CUDA_RT_CALL( hipDeviceSynchronize( ) );
        // ttm = second( );
        CUDA_RT_CALL( hipEventRecord( ttm ) );
#endif

        // 		 CUSPARSE_CALL(hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &one,
        // descra, a, ia, ja, pw, &zero, v));

#ifdef TIME_INDIVIDUAL_LIBRARY_CALLS
        // CUDA_RT_CALL( hipDeviceSynchronize( ) );
        // ttm2 = second( );
        CUDA_RT_CALL( hipEventRecord( ttm2 ) );
        CUDA_RT_CALL( hipEventSynchronize( ttm2 ) );
        CUDA_RT_CALL( hipEventElapsedTime( &ttm_ttm2, ttm, ttm2 ) );
        ttt_mv += ( ttm_ttm2 * 1e-3 );
        printf( "matvec %f (s)\n", ttm_ttm2 * 1e-3 );
#endif

        // 		 CUBLAS_CALL(hipblasDdot(cublasHandle,n, rw, 1, v, 1,&temp));
        CUBLAS_CALL(
            hipblasDotEx( cublasHandle, n, rw, HIP_R_64F, 1, v, HIP_R_64F, 1, &temp, HIP_R_64F, HIP_R_64F ) );
        alpha    = rho / temp;
        negalpha = -( alpha );
        // 		 CUBLAS_CALL(hipblasDaxpy(cublasHandle,n, &negalpha, v, 1, r, 1));

        // 		 CUBLAS_CALL(hipblasDaxpy(cublasHandle,n, &alpha,        pw, 1, x, 1));
        // 		 CUBLAS_CALL(hipblasDnrm2(cublasHandle, n, r, 1, &nrmr));
        CUBLAS_CALL(
            hipblasAxpyEx( cublasHandle, n, &negalpha, HIP_R_64F, v, HIP_R_64F, 1, r, HIP_R_64F, 1, HIP_R_64F ) );
        CUBLAS_CALL(
            hipblasAxpyEx( cublasHandle, n, &alpha, HIP_R_64F, pw, HIP_R_64F, 1, x, HIP_R_64F, 1, HIP_R_64F ) );
        CUBLAS_CALL( hipblasDnrm2( cublasHandle, n, r, 1, &nrmr ) );

        if ( nrmr < tol * nrmr0 ) {
            j = 5;
            break;
        }

        // preconditioning step (lower and upper triangular solve)
#ifdef TIME_INDIVIDUAL_LIBRARY_CALLS
        // CUDA_RT_CALL( hipDeviceSynchronize( ) );
        // ttl = second( );
        CUDA_RT_CALL( hipEventRecord( ttl ) );
#endif
        // CUSPARSE_CALL( hipsparseSetMatFillMode( descrm, HIPSPARSE_FILL_MODE_LOWER ) );
        // CUSPARSE_CALL( hipsparseSetMatDiagType( descrm, HIPSPARSE_DIAG_TYPE_UNIT ) );
        // // 		 CUSPARSE_CALL(cusparseDcsrsv_solve(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE,n,&one,
        // // descrm,vm,im,jm,info_l,r,t));
        // CUSPARSE_CALL( hipsparseDcsrsv2_solve( cusparseHandle,
        //                                       HIPSPARSE_OPERATION_NON_TRANSPOSE,
        //                                       n,
        //                                       nnz,
        //                                       &one,
        //                                       descrm,
        //                                       vm,
        //                                       im,
        //                                       jm,
        //                                       info_l,
        //                                       r,
        //                                       t,
        //                                       HIPSPARSE_SOLVE_POLICY_NO_LEVEL,
        //                                       dBuffer_l ) );
        CUSPARSE_CALL( hipsparseCreateDnVec( &vecX, n, r, HIP_R_64F ) )
        CUSPARSE_CALL( hipsparseCreateDnVec( &vecY, n, t, HIP_R_64F ) )

        // execute SpSV Lower
        CUSPARSE_CALL( hipsparseSpSV_solve( cusparseHandle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &one,
                                           descrm,
                                           vecX,
                                           vecY,
                                           HIP_R_64F,
                                           HIPSPARSE_SPSV_ALG_DEFAULT,
                                           spsvDescr_l ) )

#ifdef TIME_INDIVIDUAL_LIBRARY_CALLS
        // CUDA_RT_CALL( hipDeviceSynchronize( ) );
        // ttl2 = second( );
        CUDA_RT_CALL( hipEventRecord( ttl2 ) );
        CUDA_RT_CALL( hipEventSynchronize( ttl2 ) );
        CUDA_RT_CALL( hipEventElapsedTime( &ttl_ttl2, ttl, ttl2 ) );
        // ttu  = second( );
        CUDA_RT_CALL( hipEventRecord( ttu ) );
#endif
        // CUSPARSE_CALL( hipsparseSetMatFillMode( descrm, HIPSPARSE_FILL_MODE_UPPER ) );
        // CUSPARSE_CALL( hipsparseSetMatDiagType( descrm, HIPSPARSE_DIAG_TYPE_NON_UNIT ) );
        // // 		 CUSPARSE_CALL(cusparseDcsrsv_solve(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE,n,
        // // &one,descrm,vm,im,jm,info_u,t,s));
        // CUSPARSE_CALL( hipsparseDcsrsv2_solve( cusparseHandle,
        //                                       HIPSPARSE_OPERATION_NON_TRANSPOSE,
        //                                       n,
        //                                       nnz,
        //                                       &one,
        //                                       descrm,
        //                                       vm,
        //                                       im,
        //                                       jm,
        //                                       info_u,
        //                                       t,
        //                                       s,
        //                                       HIPSPARSE_SOLVE_POLICY_NO_LEVEL,
        //                                       dBuffer_u ) );

        CUSPARSE_CALL( hipsparseCreateDnVec( &vecX, n, t, HIP_R_64F ) )
        CUSPARSE_CALL( hipsparseCreateDnVec( &vecY, n, s, HIP_R_64F ) )

        // execute SpSV Lower
        CUSPARSE_CALL( hipsparseSpSV_solve( cusparseHandle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &one,
                                           descrm,
                                           vecX,
                                           vecY,
                                           HIP_R_64F,
                                           HIPSPARSE_SPSV_ALG_DEFAULT,
                                           spsvDescr_u ) )
#ifdef TIME_INDIVIDUAL_LIBRARY_CALLS
        // CUDA_RT_CALL( hipDeviceSynchronize( ) );
        // ttu2 = second( );
        CUDA_RT_CALL( hipEventRecord( ttu2 ) );
        CUDA_RT_CALL( hipEventSynchronize( ttu2 ) );
        CUDA_RT_CALL( hipEventElapsedTime( &ttu_ttu2, ttu, ttu2 ) );
        ttt_sv += ( ttl_ttl2 * 1e-3 ) + ( ttu_ttu2 * 1e-3 );
        printf( "solve lower %f (s), upper %f (s) \n", ttl_ttl2 * 1e-3, ttu_ttu2 * 1e-3 );
#endif
        // matrix-vector multiplication
#ifdef TIME_INDIVIDUAL_LIBRARY_CALLS
        // CUDA_RT_CALL( hipDeviceSynchronize( ) );
        // ttm = second( );
        CUDA_RT_CALL( hipEventRecord( ttm ) );
#endif

        // 		 CUSPARSE_CALL(hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &one,
        // descra, a, ia, ja, s, &zero, t));
#ifdef TIME_INDIVIDUAL_LIBRARY_CALLS
        // CUDA_RT_CALL( hipDeviceSynchronize( ) );
        // ttm2 = second( );
        CUDA_RT_CALL( hipEventRecord( ttm2 ) );
        CUDA_RT_CALL( hipEventSynchronize( ttm2 ) );
        CUDA_RT_CALL( hipEventElapsedTime( &ttm_ttm2, ttm, ttm2 ) );
        ttt_mv += ( ttm_ttm2 * 1e-3 );
        printf( "matvec %f (s)\n", ttm_ttm2 * 1e-3 );
#endif

        // 		 CUBLAS_CALL(hipblasDdot(cublasHandle,n, t, 1, r, 1,&temp));
        // 		 CUBLAS_CALL(hipblasDdot(cublasHandle,n, t, 1, t, 1,&temp2));
        CUBLAS_CALL(
            hipblasDotEx( cublasHandle, n, t, HIP_R_64F, 1, r, HIP_R_64F, 1, &temp, HIP_R_64F, HIP_R_64F ) );
        CUBLAS_CALL(
            hipblasDotEx( cublasHandle, n, t, HIP_R_64F, 1, t, HIP_R_64F, 1, &temp2, HIP_R_64F, HIP_R_64F ) );

        omega    = temp / temp2;
        negomega = -( omega );
        // 		 CUBLAS_CALL(hipblasDaxpy(cublasHandle,n, &omega, s, 1, x, 1));
        // 		 CUBLAS_CALL(hipblasDaxpy(cublasHandle,n, &negomega, t, 1, r, 1));

        // 		 CUBLAS_CALL(hipblasDnrm2(cublasHandle,n, r, 1,&nrmr));

        CUBLAS_CALL(
            hipblasAxpyEx( cublasHandle, n, &omega, HIP_R_64F, s, HIP_R_64F, 1, x, HIP_R_64F, 1, HIP_R_64F ) );
        CUBLAS_CALL(
            hipblasAxpyEx( cublasHandle, n, &negomega, HIP_R_64F, t, HIP_R_64F, 1, r, HIP_R_64F, 1, HIP_R_64F ) );
        CUBLAS_CALL( hipblasDnrm2( cublasHandle, n, r, 1, &nrmr ) );

        if ( nrmr < tol * nrmr0 ) {
            i++;
            j = 0;
            break;
        }
        i++;
    }

#ifdef TIME_INDIVIDUAL_LIBRARY_CALLS
    printf( "gpu total solve time %f (s), matvec time %f (s)\n", ttt_sv, ttt_mv );
#endif
}

int test_bicgstab( char *      matrix_filename,
                   char *      coloring_filename,
                   const char *element_type,
                   int         symmetrize,
                   int         debug,
                   double      damping,
                   int         maxit,
                   double      tol,
                   float       err,
                   float       eps ) {

    hipblasHandle_t   cublasHandle   = NULL;
    hipsparseHandle_t cusparseHandle = NULL;
    // hipsparseMatDescr_t descra         = NULL;
    // hipsparseMatDescr_t descrm         = NULL;
    hipsparseSpMatDescr_t descra      = NULL;
    hipsparseSpMatDescr_t descrm      = NULL;
    hipsparseMatDescr_t   descra_uli  = NULL;
    hipsparseDnVecDescr_t vecX        = NULL;
    hipsparseDnVecDescr_t vecY        = NULL;
    hipsparseSpSVDescr_t  spsvDescr_l = NULL;
    hipsparseSpSVDescr_t  spsvDescr_u = NULL;
    hipsparseFillMode_t   fillmode;
    hipsparseDiagType_t   diagtype;
    hipStream_t         stream = NULL;
    csrilu02Info_t       info_M = NULL;
    // csrsv2Info_t       info_l         = NULL;
    // csrsv2Info_t       info_u         = NULL;
    // hipsparseStatus_t status1, status2, status3;
    double *devPtrAval       = nullptr;
    int *   devPtrAcolsIndex = nullptr;
    int *   devPtrArowsIndex = nullptr;
    double *devPtrMval       = nullptr;
    int *   devPtrMcolsIndex = nullptr;
    int *   devPtrMrowsIndex = nullptr;
    double *devPtrX          = nullptr;
    double *devPtrF          = nullptr;
    double *devPtrR          = nullptr;
    double *devPtrRW         = nullptr;
    double *devPtrP          = nullptr;
    double *devPtrPW         = nullptr;
    double *devPtrS          = nullptr;
    double *devPtrT          = nullptr;
    double *devPtrV          = nullptr;
    double *Aval             = nullptr;
    int *   AcolsIndex       = nullptr;
    int *   ArowsIndex       = nullptr;
    double *Mval             = nullptr;
    // int *   MrowsIndex       = nullptr;
    // int *   McolsIndex       = nullptr;
    double *x  = nullptr;
    double *tx = nullptr;
    double *f  = nullptr;
    double *r  = nullptr;
    double *rw = nullptr;
    double *p  = nullptr;
    double *pw = nullptr;
    double *s  = nullptr;
    double *t  = nullptr;
    double *v  = nullptr;
    int     matrixM;
    int     matrixN;
    int     matrixSizeAval;
    int     matrixSizeAcolsIndex;
    int     matrixSizeArowsIndex;
    int     mSizeAval;
    // int     mSizeAcolsIndex;
    // int     mSizeArowsIndex;
    int arraySizeX;
    int arraySizeF;
    int arraySizeR;
    int arraySizeRW;
    int arraySizeP;
    int arraySizePW;
    int arraySizeS;
    int arraySizeT;
    int arraySizeV;
    int nnz;
    int mNNZ;
    // long long           flops;
    // double              start, stop;
    int num_iterations;
    // int                 nbrTests;
    int count;
    int base;
    int mbase;
    // hipsparseOperation_t trans;
    double alpha;
    double ttt_sv = 0.0;

    printf( "Testing %cbicgstab\n", *element_type );

    alpha = damping;
    // trans = HIPSPARSE_OPERATION_NON_TRANSPOSE;

    /* load the coefficient matrix */
    if ( loadMMSparseMatrix( matrix_filename,
                             *element_type,
                             true,
                             &matrixM,
                             &matrixN,
                             &nnz,
                             &Aval,
                             &ArowsIndex,
                             &AcolsIndex,
                             symmetrize ) ) {
        CLEANUP( );
        fprintf( stderr, "!!!! cusparseLoadMMSparseMatrix FAILED\n" );
        return EXIT_FAILURE;
    }

    matrixSizeAval       = nnz;
    matrixSizeAcolsIndex = matrixSizeAval;
    matrixSizeArowsIndex = matrixM + 1;
    base                 = ArowsIndex[0];
    if ( matrixM != matrixN ) {
        fprintf( stderr, "!!!! matrix MUST be square, error: m=%d != n=%d\n", matrixM, matrixN );
        return EXIT_FAILURE;
    }
    printf( "^^^^ M=%d, N=%d, nnz=%d\n", matrixM, matrixN, nnz );

    /* set some extra parameters for lower triangular factor */
    mNNZ      = ArowsIndex[matrixM] - ArowsIndex[0];
    mSizeAval = mNNZ;
    // mSizeAcolsIndex = mSizeAval;
    // mSizeArowsIndex = matrixM + 1;
    mbase = ArowsIndex[0];

    /* compressed sparse row */
    arraySizeX  = matrixN;
    arraySizeF  = matrixM;
    arraySizeR  = matrixM;
    arraySizeRW = matrixM;
    arraySizeP  = matrixN;
    arraySizePW = matrixN;
    arraySizeS  = matrixM;
    arraySizeT  = matrixM;
    arraySizeV  = matrixM;

    CUBLAS_CALL( hipblasCreate( &cublasHandle ) );
    CUSPARSE_CALL( hipsparseCreate( &cusparseHandle ) )

    /* create three matrix descriptors */
    // status1 = hipsparseCreateMatDescr( &descra );
    // status2 = hipsparseCreateMatDescr( &descrm );
    // if ( ( status1 != HIPSPARSE_STATUS_SUCCESS ) || ( status2 != HIPSPARSE_STATUS_SUCCESS ) ) {
    //     fprintf( stderr, "!!!! CUSPARSE hipsparseCreateMatDescr (coefficient matrix or preconditioner) error\n" );
    //     return EXIT_FAILURE;
    // }

    /* allocate device memory for csr matrix and vectors */
    CUDA_RT_CALL( hipMalloc( reinterpret_cast<void **>( &devPtrX ), sizeof( devPtrX[0] ) * arraySizeX ) );
    CUDA_RT_CALL( hipMalloc( reinterpret_cast<void **>( &devPtrF ), sizeof( devPtrF[0] ) * arraySizeF ) );
    CUDA_RT_CALL( hipMalloc( reinterpret_cast<void **>( &devPtrR ), sizeof( devPtrR[0] ) * arraySizeR ) );
    CUDA_RT_CALL( hipMalloc( reinterpret_cast<void **>( &devPtrRW ), sizeof( devPtrRW[0] ) * arraySizeRW ) );
    CUDA_RT_CALL( hipMalloc( reinterpret_cast<void **>( &devPtrP ), sizeof( devPtrP[0] ) * arraySizeP ) );
    CUDA_RT_CALL( hipMalloc( reinterpret_cast<void **>( &devPtrPW ), sizeof( devPtrPW[0] ) * arraySizePW ) );
    CUDA_RT_CALL( hipMalloc( reinterpret_cast<void **>( &devPtrS ), sizeof( devPtrS[0] ) * arraySizeS ) );
    CUDA_RT_CALL( hipMalloc( reinterpret_cast<void **>( &devPtrT ), sizeof( devPtrT[0] ) * arraySizeT ) );
    CUDA_RT_CALL( hipMalloc( reinterpret_cast<void **>( &devPtrV ), sizeof( devPtrV[0] ) * arraySizeV ) );
    CUDA_RT_CALL( hipMalloc( reinterpret_cast<void **>( &devPtrAval ), sizeof( devPtrAval[0] ) * matrixSizeAval ) );
    CUDA_RT_CALL( hipMalloc( reinterpret_cast<void **>( &devPtrAcolsIndex ),
                              sizeof( devPtrAcolsIndex[0] ) * matrixSizeAcolsIndex ) );
    CUDA_RT_CALL( hipMalloc( reinterpret_cast<void **>( &devPtrArowsIndex ),
                              sizeof( devPtrArowsIndex[0] ) * matrixSizeArowsIndex ) );
    CUDA_RT_CALL( hipMalloc( reinterpret_cast<void **>( &devPtrMval ), sizeof( devPtrMval[0] ) * mSizeAval ) );

    /* allocate host memory for  vectors */
    x    = ( double * )malloc( arraySizeX * sizeof( x[0] ) );
    f    = ( double * )malloc( arraySizeF * sizeof( f[0] ) );
    r    = ( double * )malloc( arraySizeR * sizeof( r[0] ) );
    rw   = ( double * )malloc( arraySizeRW * sizeof( rw[0] ) );
    p    = ( double * )malloc( arraySizeP * sizeof( p[0] ) );
    pw   = ( double * )malloc( arraySizePW * sizeof( pw[0] ) );
    s    = ( double * )malloc( arraySizeS * sizeof( s[0] ) );
    t    = ( double * )malloc( arraySizeT * sizeof( t[0] ) );
    v    = ( double * )malloc( arraySizeV * sizeof( v[0] ) );
    tx   = ( double * )malloc( arraySizeX * sizeof( tx[0] ) );
    Mval = ( double * )malloc( matrixSizeAval * sizeof( Mval[0] ) );
    if ( ( !Aval ) || ( !AcolsIndex ) || ( !ArowsIndex ) || ( !Mval ) || ( !x ) || ( !f ) || ( !r ) || ( !rw ) ||
         ( !p ) || ( !pw ) || ( !s ) || ( !t ) || ( !v ) || ( !tx ) ) {
        CLEANUP( );
        fprintf( stderr, "!!!! memory allocation error\n" );
        return EXIT_FAILURE;
    }
    /* use streams */
    int useStream = 0;
    if ( useStream ) {

        CUDA_RT_CALL( hipStreamCreate( &stream ) );
        CUBLAS_CALL( hipblasSetStream( cublasHandle, stream ) );
        CUSPARSE_CALL( hipsparseSetStream( cusparseHandle, stream ) );
    }

    /* clean memory */
    CUDA_RT_CALL( hipMemset( reinterpret_cast<void *>( devPtrX ), 0, sizeof( devPtrX[0] ) * arraySizeX ) );
    CUDA_RT_CALL( hipMemset( reinterpret_cast<void *>( devPtrF ), 0, sizeof( devPtrF[0] ) * arraySizeF ) );
    CUDA_RT_CALL( hipMemset( reinterpret_cast<void *>( devPtrR ), 0, sizeof( devPtrR[0] ) * arraySizeR ) );
    CUDA_RT_CALL( hipMemset( reinterpret_cast<void *>( devPtrRW ), 0, sizeof( devPtrRW[0] ) * arraySizeRW ) );
    CUDA_RT_CALL( hipMemset( reinterpret_cast<void *>( devPtrP ), 0, sizeof( devPtrP[0] ) * arraySizeP ) );
    CUDA_RT_CALL( hipMemset( reinterpret_cast<void *>( devPtrPW ), 0, sizeof( devPtrPW[0] ) * arraySizePW ) );
    CUDA_RT_CALL( hipMemset( reinterpret_cast<void *>( devPtrS ), 0, sizeof( devPtrS[0] ) * arraySizeS ) );
    CUDA_RT_CALL( hipMemset( reinterpret_cast<void *>( devPtrT ), 0, sizeof( devPtrT[0] ) * arraySizeT ) );
    CUDA_RT_CALL( hipMemset( reinterpret_cast<void *>( devPtrV ), 0, sizeof( devPtrV[0] ) * arraySizeV ) );
    CUDA_RT_CALL( hipMemset( reinterpret_cast<void *>( devPtrAval ), 0, sizeof( devPtrAval[0] ) * matrixSizeAval ) );
    CUDA_RT_CALL( hipMemset(
        reinterpret_cast<void *>( devPtrAcolsIndex ), 0, sizeof( devPtrAcolsIndex[0] ) * matrixSizeAcolsIndex ) );
    CUDA_RT_CALL( hipMemset(
        reinterpret_cast<void *>( devPtrArowsIndex ), 0, sizeof( devPtrArowsIndex[0] ) * matrixSizeArowsIndex ) );
    CUDA_RT_CALL( hipMemset( reinterpret_cast<void *>( devPtrMval ), 0, sizeof( devPtrMval[0] ) * mSizeAval ) );

    memset( x, 0, arraySizeX * sizeof( x[0] ) );
    memset( f, 0, arraySizeF * sizeof( f[0] ) );
    memset( r, 0, arraySizeR * sizeof( r[0] ) );
    memset( rw, 0, arraySizeRW * sizeof( rw[0] ) );
    memset( p, 0, arraySizeP * sizeof( p[0] ) );
    memset( pw, 0, arraySizePW * sizeof( pw[0] ) );
    memset( s, 0, arraySizeS * sizeof( s[0] ) );
    memset( t, 0, arraySizeT * sizeof( t[0] ) );
    memset( v, 0, arraySizeV * sizeof( v[0] ) );
    memset( tx, 0, arraySizeX * sizeof( tx[0] ) );

    // Timing
    hipEvent_t start_matrix_copy { nullptr };
    hipEvent_t stop_matrix_copy { nullptr };
    hipEvent_t ttm { nullptr };
    hipEvent_t ttm2 { nullptr };
    float       ttm_ttm2 {};
    hipEvent_t ttl { nullptr };
    hipEvent_t ttl2 { nullptr };
    float       ttl_ttl2 {};
    hipEvent_t ttu { nullptr };
    hipEvent_t ttu2 { nullptr };
    float       ttu_ttu2 {};
    hipEvent_t start_ilu { nullptr };
    hipEvent_t stop_ilu { nullptr };
    hipEvent_t start_event { nullptr };
    hipEvent_t stop_event { nullptr };
    float       elapsed_events_ms {};

    CUDA_RT_CALL( hipEventCreate( &start_matrix_copy, hipEventBlockingSync ) );
    CUDA_RT_CALL( hipEventCreate( &stop_matrix_copy, hipEventBlockingSync ) );
    CUDA_RT_CALL( hipEventCreate( &ttm, hipEventBlockingSync ) );
    CUDA_RT_CALL( hipEventCreate( &ttm2, hipEventBlockingSync ) );
    CUDA_RT_CALL( hipEventCreate( &ttl, hipEventBlockingSync ) );
    CUDA_RT_CALL( hipEventCreate( &ttl2, hipEventBlockingSync ) );
    CUDA_RT_CALL( hipEventCreate( &ttu, hipEventBlockingSync ) );
    CUDA_RT_CALL( hipEventCreate( &ttu2, hipEventBlockingSync ) );
    CUDA_RT_CALL( hipEventCreate( &start_ilu, hipEventBlockingSync ) );
    CUDA_RT_CALL( hipEventCreate( &stop_ilu, hipEventBlockingSync ) );
    CUDA_RT_CALL( hipEventCreate( &start_event, hipEventBlockingSync ) );
    CUDA_RT_CALL( hipEventCreate( &stop_event, hipEventBlockingSync ) );

    // Moved here from ILU section to prevent runtime error
    devPtrMrowsIndex = devPtrArowsIndex;
    devPtrMcolsIndex = devPtrAcolsIndex;

    /* create the test matrix and vectors on the host */
    if ( base ) {
        CUSPARSE_CALL( hipsparseCreateCsr( &descra,
                                          matrixM,
                                          matrixN,
                                          nnz,
                                          devPtrArowsIndex,
                                          devPtrAcolsIndex,
                                          devPtrAval,
                                          HIPSPARSE_INDEX_32I,
                                          HIPSPARSE_INDEX_32I,
                                          HIPSPARSE_INDEX_BASE_ONE,
                                          HIP_R_64F ) )
    } else {
        CUSPARSE_CALL( hipsparseCreateCsr( &descra,
                                          matrixM,
                                          matrixN,
                                          nnz,
                                          devPtrArowsIndex,
                                          devPtrAcolsIndex,
                                          devPtrAval,
                                          HIPSPARSE_INDEX_32I,
                                          HIPSPARSE_INDEX_32I,
                                          HIPSPARSE_INDEX_BASE_ZERO,
                                          HIP_R_64F ) )
    }
    if ( mbase ) {
        CUSPARSE_CALL( hipsparseCreateCsr( &descrm,
                                          matrixM,
                                          matrixN,
                                          nnz,
                                          devPtrMrowsIndex,
                                          devPtrMcolsIndex,
                                          devPtrMval,
                                          HIPSPARSE_INDEX_32I,
                                          HIPSPARSE_INDEX_32I,
                                          HIPSPARSE_INDEX_BASE_ONE,
                                          HIP_R_64F ) )
    } else {
        CUSPARSE_CALL( hipsparseCreateCsr( &descrm,
                                          matrixM,
                                          matrixN,
                                          nnz,
                                          devPtrMrowsIndex,
                                          devPtrMcolsIndex,
                                          devPtrMval,
                                          HIPSPARSE_INDEX_32I,
                                          HIPSPARSE_INDEX_32I,
                                          HIPSPARSE_INDEX_BASE_ZERO,
                                          HIP_R_64F ) )
    }

    // compute the right-hand-side f=A*e, where e=[1, ..., 1]'
    for ( int i = 0; i < arraySizeP; i++ ) {
        p[i] = 1.0;
    }

    /* copy the csr matrix and vectors into device memory */
    CUDA_RT_CALL( hipEventRecord( start_matrix_copy ) );

    CUDA_RT_CALL(
        hipMemcpy( devPtrAval, Aval, ( size_t )( matrixSizeAval * sizeof( Aval[0] ) ), hipMemcpyHostToDevice ) );
    CUDA_RT_CALL( hipMemcpy( devPtrAcolsIndex,
                              AcolsIndex,
                              ( size_t )( matrixSizeAcolsIndex * sizeof( AcolsIndex[0] ) ),
                              hipMemcpyHostToDevice ) );
    CUDA_RT_CALL( hipMemcpy( devPtrArowsIndex,
                              ArowsIndex,
                              ( size_t )( matrixSizeArowsIndex * sizeof( ArowsIndex[0] ) ),
                              hipMemcpyHostToDevice ) );
    CUDA_RT_CALL( hipMemcpy(
        devPtrMval, devPtrAval, ( size_t )( matrixSizeAval * sizeof( devPtrMval[0] ) ), hipMemcpyDeviceToDevice ) );

    CUDA_RT_CALL( hipEventRecord( stop_matrix_copy ) );
    CUDA_RT_CALL( hipEventSynchronize( stop_matrix_copy ) );
    CUDA_RT_CALL( hipEventElapsedTime( &elapsed_events_ms, start_matrix_copy, stop_matrix_copy ) );

    fprintf( stdout, "Copy matrix from CPU to GPU, time(s) = %f\n", elapsed_events_ms * 1e-3 );

    CUDA_RT_CALL( hipMemcpy( devPtrX, x, ( size_t )( arraySizeX * sizeof( devPtrX[0] ) ), hipMemcpyHostToDevice ) );
    CUDA_RT_CALL( hipMemcpy( devPtrF, f, ( size_t )( arraySizeF * sizeof( devPtrF[0] ) ), hipMemcpyHostToDevice ) );
    CUDA_RT_CALL( hipMemcpy( devPtrR, r, ( size_t )( arraySizeR * sizeof( devPtrR[0] ) ), hipMemcpyHostToDevice ) );
    CUDA_RT_CALL(
        hipMemcpy( devPtrRW, rw, ( size_t )( arraySizeRW * sizeof( devPtrRW[0] ) ), hipMemcpyHostToDevice ) );
    CUDA_RT_CALL( hipMemcpy( devPtrP, p, ( size_t )( arraySizeP * sizeof( devPtrP[0] ) ), hipMemcpyHostToDevice ) );
    CUDA_RT_CALL(
        hipMemcpy( devPtrPW, pw, ( size_t )( arraySizePW * sizeof( devPtrPW[0] ) ), hipMemcpyHostToDevice ) );
    CUDA_RT_CALL( hipMemcpy( devPtrS, s, ( size_t )( arraySizeS * sizeof( devPtrS[0] ) ), hipMemcpyHostToDevice ) );
    CUDA_RT_CALL( hipMemcpy( devPtrT, t, ( size_t )( arraySizeT * sizeof( devPtrT[0] ) ), hipMemcpyHostToDevice ) );
    CUDA_RT_CALL( hipMemcpy( devPtrV, v, ( size_t )( arraySizeV * sizeof( devPtrV[0] ) ), hipMemcpyHostToDevice ) );

    /* --- GPU --- */
    /* create the analysis info (for lower and upper triangular factors) */
    size_t bufferSize_l = 0;
    size_t bufferSize_u = 0;

    void * dBuffer    = NULL;
    size_t bufferSize = 0;

    /* Calculate LOWER buffersize */
    // Create dense vector X_u
    CUSPARSE_CALL( hipsparseCreateDnVec( &vecX, matrixM, devPtrP, HIP_R_64F ) );

    // Create dense vector Y_u
    CUSPARSE_CALL( hipsparseCreateDnVec( &vecY, matrixM, devPtrT, HIP_R_64F ) );

    // Create opaque data structure, that holds analysis data between calls.
    CUSPARSE_CALL( hipsparseSpSV_createDescr( &spsvDescr_l ) );

    // Specify Lower|Upper fill mode.
    fillmode = HIPSPARSE_FILL_MODE_LOWER;
    CUSPARSE_CALL( hipsparseSpMatSetAttribute( descrm, HIPSPARSE_SPMAT_FILL_MODE, &fillmode, sizeof( fillmode ) ) );
    // Specify Unit|Non-Unit diagonal type.
    diagtype = HIPSPARSE_DIAG_TYPE_UNIT;
    CUSPARSE_CALL( hipsparseSpMatSetAttribute( descrm, HIPSPARSE_SPMAT_DIAG_TYPE, &diagtype, sizeof( diagtype ) ) );

    // allocate an external buffer for analysis
    CUSPARSE_CALL( hipsparseSpSV_bufferSize( cusparseHandle,
                                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                            &alpha,
                                            descrm,
                                            vecX,
                                            vecY,
                                            HIP_R_64F,
                                            HIPSPARSE_SPSV_ALG_DEFAULT,
                                            spsvDescr_l,
                                            &bufferSize_l ) );

    /* Calculate UPPER buffersize */
    // Create dense vector X_u
    CUSPARSE_CALL( hipsparseCreateDnVec( &vecX, matrixM, devPtrT, HIP_R_64F ) );

    // Create dense vector Y_u
    CUSPARSE_CALL( hipsparseCreateDnVec( &vecY, matrixM, devPtrPW, HIP_R_64F ) );

    // Create opaque data structure, that holds analysis data between calls.
    CUSPARSE_CALL( hipsparseSpSV_createDescr( &spsvDescr_u ) );

    // Specify Lower|Upper fill mode.
    fillmode = HIPSPARSE_FILL_MODE_UPPER;
    CUSPARSE_CALL( hipsparseSpMatSetAttribute( descrm, HIPSPARSE_SPMAT_FILL_MODE, &fillmode, sizeof( fillmode ) ) );
    // Specify Unit|Non-Unit diagonal type.
    diagtype = HIPSPARSE_DIAG_TYPE_NON_UNIT;
    CUSPARSE_CALL( hipsparseSpMatSetAttribute( descrm, HIPSPARSE_SPMAT_DIAG_TYPE, &diagtype, sizeof( diagtype ) ) );

    // allocate an external buffer for analysis
    CUSPARSE_CALL( hipsparseSpSV_bufferSize( cusparseHandle,
                                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                            &alpha,
                                            descrm,
                                            vecX,
                                            vecY,
                                            HIP_R_64F,
                                            HIPSPARSE_SPSV_ALG_DEFAULT,
                                            spsvDescr_u,
                                            &bufferSize_u ) );

    /* Allocate max buffersize */
    bufferSize = max( bufferSize_l, bufferSize_u );
    CUDA_RT_CALL( hipMalloc( &dBuffer, bufferSize ) );

    /* analyse the lower and upper triangular factors */
    // double ttl = second( );
    CUDA_RT_CALL( hipEventRecord( ttl ) );

    // CUSPARSE_CALL( hipsparseSetMatFillMode( descrm, HIPSPARSE_FILL_MODE_LOWER ) );
    // CUSPARSE_CALL( hipsparseSetMatDiagType( descrm, HIPSPARSE_DIAG_TYPE_UNIT ) );
    // checkCudaErrors( cusparseDcsrsv_analysis( cusparseHandle,
    //                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
    //                                           matrixM,
    //                                           nnz,
    //                                           descrm,
    //                                           devPtrAval,
    //                                           devPtrArowsIndex,
    //                                           devPtrAcolsIndex,
    //                                           info_l ) );

    CUSPARSE_CALL( hipsparseCreateDnVec( &vecX, matrixM, devPtrP, HIP_R_64F ) );
    CUSPARSE_CALL( hipsparseCreateDnVec( &vecY, matrixM, devPtrT, HIP_R_64F ) );
    CUSPARSE_CALL( hipsparseSpSV_analysis( cusparseHandle,
                                          HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                          &alpha,
                                          descrm,
                                          vecX,
                                          vecY,
                                          HIP_R_64F,
                                          HIPSPARSE_SPSV_ALG_DEFAULT,
                                          spsvDescr_l,
                                          dBuffer ) );

    // CUDA_RT_CALL( hipDeviceSynchronize( ) );
    // double ttl2 = second( );
    CUDA_RT_CALL( hipEventRecord( ttl2 ) );
    CUDA_RT_CALL( hipEventSynchronize( ttl2 ) );
    // CUDA_RT_CALL( hipEventElapsedTime( &elapsed_events_ms, start_event, stop_event ) );

    // double ttu = second( );
    CUDA_RT_CALL( hipEventRecord( ttu ) );

    // CUSPARSE_CALL( hipsparseSetMatFillMode( descrm, HIPSPARSE_FILL_MODE_UPPER ) );
    // CUSPARSE_CALL( hipsparseSetMatDiagType( descrm, HIPSPARSE_DIAG_TYPE_NON_UNIT ) );
    // CUSPARSE_CALL( cusparseDcsrsv_analysis( cusparseHandle,
    //                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
    //                                         matrixM,
    //                                         nnz,
    //                                         descrm,
    //                                         devPtrAval,
    //                                         devPtrArowsIndex,
    //                                         devPtrAcolsIndex,
    //                                         info_u ) );

    CUSPARSE_CALL( hipsparseCreateDnVec( &vecX, matrixM, devPtrT, HIP_R_64F ) );
    CUSPARSE_CALL( hipsparseCreateDnVec( &vecY, matrixM, devPtrPW, HIP_R_64F ) );
    CUSPARSE_CALL( hipsparseSpSV_analysis( cusparseHandle,
                                          HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                          &alpha,
                                          descrm,
                                          vecX,
                                          vecY,
                                          HIP_R_64F,
                                          HIPSPARSE_SPSV_ALG_DEFAULT,
                                          spsvDescr_u,
                                          dBuffer ) );

    // CUDA_RT_CALL( hipDeviceSynchronize( ) );
    // double ttu2 = second( );

    CUDA_RT_CALL( hipEventRecord( ttu2 ) );
    CUDA_RT_CALL( hipEventSynchronize( ttu2 ) );
    CUDA_RT_CALL( hipEventElapsedTime( &ttl_ttl2, ttl, ttl2 ) );
    CUDA_RT_CALL( hipEventElapsedTime( &ttu_ttu2, ttu, ttu2 ) );

    ttt_sv += ( ttl_ttl2 * 1e-3 ) + ( ttu_ttu2 * 1e-3 );
    printf( "analysis lower %f (s), upper %f (s) \n", ttl_ttl2 * 1e-3, ttu_ttu2 * 1e-3 );

    /* compute the lower and upper triangular factors using CUSPARSE csrilu0 routine (on the GPU) */
    // double start_ilu, stop_ilu;
    printf( "CUSPARSE csrilu0\n" );
    // start_ilu = second( );
    CUDA_RT_CALL( hipEventRecord( start_ilu ) );
    // devPtrMrowsIndex = devPtrArowsIndex;
    // devPtrMcolsIndex = devPtrAcolsIndex;

    void *dBuffer_ilu    = NULL;
    int   bufferSize_ilu = 0;

    // int m = matrixM + matrixN;

    CUSPARSE_CALL( hipsparseCreateCsrilu02Info( &info_M ) );
    CUSPARSE_CALL( hipsparseCreateMatDescr( &descra_uli ) );
    CUSPARSE_CALL( hipsparseSetMatType( descra_uli, HIPSPARSE_MATRIX_TYPE_GENERAL ) );
    if ( base ) {
        CUSPARSE_CALL( hipsparseSetMatIndexBase( descra_uli, HIPSPARSE_INDEX_BASE_ONE ) );
    } else {
        // CUSPARSE_CALL( hipsparseSetMatIndexBase( descra_uli, HIPSPARSE_INDEX_BASE_ZERO ) );
    }

    CUSPARSE_CALL( hipsparseDcsrilu02_bufferSize( cusparseHandle,
                                                 matrixM,
                                                 nnz,
                                                 descra_uli,
                                                 devPtrMval,
                                                 devPtrArowsIndex,
                                                 devPtrAcolsIndex,
                                                 info_M,
                                                 &bufferSize_ilu ) );

    CUDA_RT_CALL( hipMalloc( &dBuffer_ilu, bufferSize_ilu ) );

    CUSPARSE_CALL( hipsparseDcsrilu02_analysis( cusparseHandle,
                                               matrixM,
                                               nnz,
                                               descra_uli,
                                               devPtrMval,
                                               devPtrArowsIndex,
                                               devPtrAcolsIndex,
                                               info_M,
                                               HIPSPARSE_SOLVE_POLICY_USE_LEVEL,
                                               dBuffer_ilu ) );

    // FIXME -- BUG?
    CUSPARSE_CALL( hipsparseDcsrilu02( cusparseHandle,
                                      matrixM,
                                      nnz,
                                      descra_uli,
                                      devPtrMval,
                                      devPtrArowsIndex,
                                      devPtrAcolsIndex,
                                      info_M,
                                      HIPSPARSE_SOLVE_POLICY_USE_LEVEL,
                                      dBuffer_ilu ) );

    // OLD
    // checkCudaErrors( cusparseDcsrilu0( cusparseHandle,
    //                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
    //                                        matrixM,
    //                                        descra,
    //                                        devPtrMval,
    //                                        devPtrArowsIndex,
    //                                        devPtrAcolsIndex,
    //                                        info_l ) );

    // CUDA_RT_CALL( hipDeviceSynchronize( ) );
    // stop_ilu = second( );
    CUDA_RT_CALL( hipEventRecord( stop_ilu ) );
    CUDA_RT_CALL( hipEventSynchronize( stop_ilu ) );
    CUDA_RT_CALL( hipEventElapsedTime( &elapsed_events_ms, start_ilu, stop_ilu ) );

    fprintf( stdout, "time(s) = %10.8f \n", elapsed_events_ms * 1e-3 );

    /* run the test */
    // Note that multiple iterations don't provide correct results
    // Because of inplace writes.
    num_iterations = 1;  // 10;
    CUDA_RT_CALL( hipEventRecord( start_event ) );
    for ( count = 0; count < num_iterations; count++ ) {

        gpu_pbicgstab( cublasHandle,
                       cusparseHandle,
                       matrixM,
                       matrixN,
                       nnz,
                       descra,
                       devPtrAval,
                       devPtrArowsIndex,
                       devPtrAcolsIndex,
                       descrm,
                       devPtrMval,
                       devPtrMrowsIndex,
                       devPtrMcolsIndex,
                       descra_uli,
                       vecX,
                       vecY,
                       spsvDescr_l,
                       spsvDescr_u,
                       fillmode,
                       diagtype,
                       info_M,
                       //    info_l,
                       //    info_u,
                       dBuffer,
                       dBuffer_ilu,
                       devPtrF,
                       devPtrR,
                       devPtrRW,
                       devPtrP,
                       devPtrPW,
                       devPtrS,
                       devPtrT,
                       devPtrV,
                       devPtrX,
                       maxit,
                       tol,
                       ttt_sv,
                       ttm,
                       ttm2,
                       ttm_ttm2,
                       ttl,
                       ttl2,
                       ttl_ttl2,
                       ttu,
                       ttu2,
                       ttu_ttu2 );

        CUDA_RT_CALL( hipDeviceSynchronize( ) );
    }
    // stop = second( ) / num_iterations;
    CUDA_RT_CALL( hipEventRecord( stop_event ) );
    CUDA_RT_CALL( hipEventSynchronize( stop_event ) );
    CUDA_RT_CALL( hipEventElapsedTime( &elapsed_events_ms, start_event, stop_event ) );

    fprintf( stdout, "Average bicgstab time(s) = %10.8f \n", ( elapsed_events_ms / num_iterations ) * 1e-3 );

    /* copy the result into host memory */
    CUDA_RT_CALL( hipMemcpy( tx, devPtrX, ( size_t )( arraySizeX * sizeof( tx[0] ) ), hipMemcpyDeviceToHost ) );

    /* destroy the analysis info (for lower and upper triangular factors) */
    // CUSPARSE_CALL( hipsparseDestroyCsrsv2Info( info_l ) );
    // CUSPARSE_CALL( hipsparseDestroyCsrsv2Info( info_u ) );
    CUSPARSE_CALL( hipsparseDestroyCsrilu02Info( info_M ) );

    // CUSPARSE_CALL( hipsparseDestroyMatDescr( descra ) );
    // CUSPARSE_CALL( hipsparseDestroyMatDescr( descrm ) );
    CUSPARSE_CALL( hipsparseDestroyMatDescr( descra_uli ) );

    CUSPARSE_CALL( hipsparseDestroy( cusparseHandle ) );
    CUBLAS_CALL( hipblasDestroy( cublasHandle ) );

    // CUDA_RT_CALL( hipStreamDestroy( stream ) );

    CUDA_RT_CALL( hipEventDestroy( start_matrix_copy ) );
    CUDA_RT_CALL( hipEventDestroy( stop_matrix_copy ) );
    CUDA_RT_CALL( hipEventDestroy( ttl ) );
    CUDA_RT_CALL( hipEventDestroy( ttl2 ) );
    CUDA_RT_CALL( hipEventDestroy( ttu ) );
    CUDA_RT_CALL( hipEventDestroy( ttu2 ) );
    CUDA_RT_CALL( hipEventDestroy( ttm ) );
    CUDA_RT_CALL( hipEventDestroy( ttm2 ) );
    CUDA_RT_CALL( hipEventDestroy( start_ilu ) );
    CUDA_RT_CALL( hipEventDestroy( stop_ilu ) );
    CUDA_RT_CALL( hipEventDestroy( start_event ) );
    CUDA_RT_CALL( hipEventDestroy( stop_event ) );

    return EXIT_SUCCESS;
}

int main( int argc, char *argv[] ) {
    int   status            = EXIT_FAILURE;
    char *matrix_filename   = NULL;
    char *coloring_filename = NULL;

    int    symmetrize = 0;
    int    debug      = 0;
    int    maxit      = 2000;  // 5; //2000; //1000;  //50; //5; //50; //100; //500; //10000;
    double tol = 0.0000001;    // 0.000001; //0.00001; //0.00000001; //0.0001; //0.001; //0.00000001; //0.1; //0.001;
                               // //0.00000001;
    double damping = 0.75;

    /* WARNING: it is assumed that the matrices are stores in Matrix Market format */
    printf( "WARNING: it is assumed that the matrices are stores in Matrix Market format with double as elementtype\n "
            "Usage: ./BiCGStab -F[matrix.mtx] [-E] [-D]\n" );

    printf( "Starting [%s]\n", argv[0] );
    int i         = 0;
    int temp_argc = argc;
    while ( argc ) {
        if ( *argv[i] == '-' ) {
            switch ( *( argv[i] + 1 ) ) {
            case 'F':
                matrix_filename = argv[i] + 2;
                break;
            case 'E':
                symmetrize = 1;
                break;
            case 'D':
                debug = 1;
                break;
            case 'C':
                coloring_filename = argv[i] + 2;
                break;
            default:
                fprintf( stderr, "Unknown switch '-%s'\n", argv[i] + 1 );
                return status;
            }
        }
        argc--;
        i++;
    }

    argc = temp_argc;

    // Use default input file
    if ( matrix_filename == NULL ) {
        printf( "argv[0] = %s\n", argv[0] );
        matrix_filename = sdkFindFilePath( "gr_900_900_cfg.mtx", argv[0] );

        if ( matrix_filename != NULL ) {
            printf( "Using default input file [%s]\n", matrix_filename );
        } else {
            printf( "Could not find input file = %s\n", matrix_filename );
            return EXIT_FAILURE;
        }
    } else {
        printf( "Using input file [%s]\n", matrix_filename );
    }

    //  findCudaDevice(argc, (const char **)argv);

    status = test_bicgstab( matrix_filename,
                            coloring_filename,
                            "d",
                            symmetrize,
                            debug,
                            damping,
                            maxit,
                            tol,
                            DBICGSTAB_MAX_ULP_ERR,
                            DBICGSTAB_EPS );

    return status;
}
